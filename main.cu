
#include <hip/hip_runtime.h>
#include <iostream>

static void _check_cuda_error(hipError_t error_code, char const *const func,
                              const char *const file, int const line) {
    if (!error_code) {
        return;
    }

    std::cerr << "CUDA error at " << file << ": " << line << " '" << func << "'\n";
    auto error_str = hipGetErrorString(error_code);
    std::cerr << "CUDA error " << static_cast<unsigned int>(error_code) << ": " << error_str
              << "\n";

    // Make sure we call CUDA Device Reset before exiting
    hipDeviceReset();
    exit(1);
}

// limited version of checkCudaErrors from helper_cuda.h in CUDA examples
#define CHECK_CUDA_ERROR(val) _check_cuda_error((val), #val, __FILE__, __LINE__)

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

int main() {
    printf("hello from CPU\n");

    cuda_hello<<<3,3>>>();
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    return 0;
}
